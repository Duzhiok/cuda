#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h> // For the CUDA runtime routines (prefixed with "cuda_")

#include <time.h> 


#define DATA_SIZE 9  
#define THREAD_NUM 5  
using namespace std;
int anData[DATA_SIZE];

//Generate random data
void GenerateNumber(int *pnNumber, int nSize)
{

	for (int i = 0; i < nSize; i++)  pnNumber[i] = rand() % 10;
}

//Global function to calculate square kernel: call on a host computer, performed on the equipment
__global__ static void sumofSquares(int *pnNum, int* pnResult, clock_t *pclock_ttime)
{
	int tid = threadIdx.x;
	int nSum = 0;
	int i;

	int nSize = 0;


	if (DATA_SIZE % THREAD_NUM) nSize = DATA_SIZE / THREAD_NUM + 1;
	else nSize = DATA_SIZE / THREAD_NUM; //nSize, One thread to calculate the amount of data



										 //start counting
	clock_t clock_tstart;
	if (tid == 0)  clock_tstart = clock(); 

	for (i = nSize * tid; i < (tid + 1) *nSize; i++)
	{
		nSum += (pnNum[i] * pnNum[i]);

	}
	pnResult[tid] = nSum;
	//end
	if (tid == 0) *pclock_ttime = clock() - clock_tstart; //Time with a thread
}

int main(void)
{
	GenerateNumber(anData, DATA_SIZE); //Generate random data
	int *pnGpuData, *pnResult;
	clock_t *pclock_ttime;//time storage
	int *nSummat;
	hipHostMalloc((void**)&nSummat, sizeof(int) * THREAD_NUM, hipHostMallocDefault);
	hipMalloc((void**)&pnGpuData, sizeof(int) * DATA_SIZE);
	hipMalloc((void**)&pnResult, sizeof(int) * THREAD_NUM);
	hipMalloc((void**)&pclock_ttime, sizeof(clock_t));

	hipMemcpy(pnGpuData, anData, sizeof(int)*DATA_SIZE, hipMemcpyHostToDevice);



	sumofSquares <<< 1, THREAD_NUM, 0 >>>(pnGpuData, pnResult, pclock_ttime);

	hipMemcpy(nSummat, pnResult, sizeof(int) * THREAD_NUM, hipMemcpyDeviceToHost);
	clock_t pclocksum;
	hipMemcpy(&pclocksum, pclock_ttime, sizeof(clock_t), hipMemcpyDeviceToHost);

	//operation of the CPU
	int finishsum = 0;
	for (size_t i = 0; i < THREAD_NUM; i++)
	{
		finishsum = finishsum + nSummat[i];

	}

	printf("SuM = %d    Time = %d\n", finishsum, pclocksum);
	hipFree(pnGpuData);
	hipFree(pnResult);
	hipFree(pclock_ttime);
	system("pause");
	//return 0;

}